#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>


// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = index; i < n; i+=stride)
    {
        y[i] = x[i] + y[i];
    } 
}

int main(void){
    int N = 1<<29; // 1M elements
    int blockSize = 256;
    int numBlocks = (N+blockSize-1) / blockSize;

    std::cout << "Calculating vector addition of " << N << " elements using "<< blockSize << " threads for each block, and " << numBlocks << " blocks."<< std::endl; 

    // Allocate Unified Memory -- accessible from CPU or GPU
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
   
 // initialize x and y arrays on the host
    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
   
    int deviceID=0;
    hipMemPrefetchAsync((void *)x, N*sizeof(float), deviceID);
    hipMemPrefetchAsync((void *)y, N*sizeof(float), deviceID);

    // run kernal on 1M elements on the CPU   
    add<<<numBlocks,blockSize>>>(N, x, y); 

    // Wait for GPU to finish.
    hipDeviceSynchronize();

    // check for errors (all values should be 3.0f)
    // Check for errors
    float maxError = 0.0f;
    for(int i=0; i<N; i++) {
        maxError=fmax(maxError, fabs(y[i]-3.0f));
    }

    std::cout << "Max Error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}
