
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = index; i < n; i+=stride)
    {
        y[i] = x[i] + y[i];
    } 
}

int main(void)
{
    int N = 1<<29; // 1M elements
    int blockSize = 256;
    int numBlocks = (N+blockSize-1) / blockSize;

    std::cout << "Calculating vector addition of " << N << " elements using "<< blockSize << " threads for each block and " << numBlocks << " blocks."<< std::endl; 

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
   
 // initialize x and y arrays on the host
    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // run kernal on 1M elements.
    add<<<numBlocks,blockSize>>>(N, x, y);

    // Wait for GPU to finish.
    hipDeviceSynchronize();

    // Check for errors
    float maxError = 0.0f;
    for(int i=0; i<N; i++) {
        maxError=fmax(maxError, fabs(y[i]-3.0f));
    }

    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
